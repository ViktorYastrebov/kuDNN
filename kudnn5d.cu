#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include <math.h>
#include "kudnn.h"
#include <limits.h>


__global__ void krnlXCorr5d(double *src, int N, int C, int H, int W, int D,
                            double *flt, int Kw, int Cw, int Hf, int Wf, int Df,
                            double *dst, int Ny, int K, int Hy, int Wy, int Dy,
                            int hpad, int wpad, int dpad){
    int i = threadIdx.x, k = threadIdx.y; 
    int h = blockIdx.x, w = blockIdx.y, d = blockIdx.z;
    int j,l,m,n;
    int hsrc, wsrc, dsrc;

    double sum=0;
    for(j=0;j<C;j++){ 
        for(l=0; l<Hf;l++){
        for(m=0; m<Wf;m++){
        for(n=0; n<Df;n++){
            hsrc = h+l-hpad; wsrc = w+m-wpad; dsrc = d+n-dpad;
            if(hsrc >= 0 && wsrc >= 0 &&  dsrc >= 0 &&  hsrc < H && wsrc < W && dsrc < D) 
                sum += src[ind5d(C,H,W,D,i,j,hsrc,wsrc,dsrc)] * flt[ind5d(C,Hf,Wf,Df,k,j,l,m,n)];
        }}}
    }
    dst[ind5d(K,Hy,Wy,Dy,i,k,h,w,d)] = sum;
}


hipdnnStatus_t CUDNNWINAPI kunetConvolutionForward(        hipdnnHandle_t                     handle,
                                                          const void                         *alpha,
                                                          const hipdnnTensorDescriptor_t       srcDesc,
                                                          const void                         *srcData,
                                                          const hipdnnFilterDescriptor_t       filterDesc,
                                                          const void                         *filterData,
                                                          const hipdnnConvolutionDescriptor_t  convDesc,
                                                          hipdnnConvolutionFwdAlgo_t           algo,
                                                          void                               *workSpace,
                                                          size_t                              workSpaceSizeInBytes,            
                                                          const void                         *beta,
                                                          const hipdnnTensorDescriptor_t       destDesc,
                                                          void                               *destData
                                                 ){
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;
    hipdnnDataType_t dataType; // image data type
    hipdnnConvolutionMode_t mode;
    int ndimsreq=5; int convndimsreq=3;

    int xndims, xDims[ndimsreq], xStrides[ndimsreq];
    hipdnnGetTensorNdDescriptor(srcDesc,  ndimsreq, &dataType, &xndims, xDims, xStrides);
    assert(dataType == HIPDNN_DATA_DOUBLE);

    int wndims, wDims[ndimsreq];
    hipdnnGetFilterNdDescriptor(filterDesc, ndimsreq, &dataType, &wndims, wDims);
    assert(dataType == HIPDNN_DATA_DOUBLE);
    printf("%d %d\n", xndims, wndims);
    assert(xndims == wndims);

    int yndims, yDims[ndimsreq], yStrides[ndimsreq];
    hipdnnGetTensorNdDescriptor(destDesc,  ndimsreq, &dataType, &yndims, yDims, yStrides);
    assert(dataType == HIPDNN_DATA_DOUBLE);
    assert(xndims == yndims);

    int convndims, convPad[convndimsreq], convStride[convndimsreq], convUpscale[convndimsreq];
    cudnnGetConvolutionNdDescriptor(convDesc, convndimsreq, &convndims, convPad, convStride, convUpscale, &mode);
    assert(convndims==(xndims-2));

    printf("N:%d C:%d H:%d W:%d D:%d\n",        cat5d(xDims));
    printf("K:%d C:%d Hw:%d Ww:%d Dw:%d\n",     cat5d(wDims));
    printf("N:%d K:%d Hy:%d Wy:%d Dy:%d\n",     cat5d(yDims));
    printf("\n");


    dim3 threads(   yDims[0],   yDims[1],   1); // N K
    dim3 grid(      yDims[2],   yDims[3],   yDims[4]); // Hy Wy Dy
    if(mode == HIPDNN_CROSS_CORRELATION){
        // xcorr(x,w)
        krnlXCorr5d<<<grid, threads>>>((double *)srcData, cat5d(xDims),
                            (double *)filterData, cat5d(wDims),
                            (double *)destData, cat5d(yDims), convPad[0], convPad[1], convPad[2]);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

    }else if(mode == HIPDNN_CONVOLUTION){
        // conv(x,w)
        status = HIPDNN_STATUS_NOT_SUPPORTED;
        /*krnlConv4d<<<grid,threads>>>((double *)srcData, N, C, H, W,
                                    (double *)filterData, Hf, Wf, K,
                                    (double *)destData, Ho, Wo, pad_h, pad_w);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );*/
        //status = HIPDNN_STATUS_NOT_SUPPORTED;
    }else{
        status = HIPDNN_STATUS_BAD_PARAM;
    }
    return status;
}

hipdnnStatus_t CUDNNWINAPI kunetConvolutionBackwardFilter( hipdnnHandle_t                       handle,
                                                          const void                         *alpha,
                                                          const hipdnnTensorDescriptor_t       srcDesc,
                                                          const void                         *srcData,
                                                          const hipdnnTensorDescriptor_t       diffDesc,
                                                          const void                         *diffData,
                                                          const hipdnnConvolutionDescriptor_t  convDesc,
                                                          const void                         *beta,
                                                          const hipdnnFilterDescriptor_t       gradDesc,
                                                          void                               *gradData
                                                        ){
    return HIPDNN_STATUS_NOT_SUPPORTED;
}

hipdnnStatus_t CUDNNWINAPI kunetConvolutionBackwardData(  hipdnnHandle_t                       handle,
                                                         const void                         *alpha,
                                                         const hipdnnFilterDescriptor_t       filterDesc,
                                                         const void                         *filterData,
                                                         const hipdnnTensorDescriptor_t       diffDesc,
                                                         const void                         *diffData,
                                                         const hipdnnConvolutionDescriptor_t  convDesc,
                                                         const void                         *beta,
                                                         const hipdnnTensorDescriptor_t       gradDesc,
                                                         void                               *gradData
                                                       ){

    return HIPDNN_STATUS_NOT_SUPPORTED;
}

hipdnnStatus_t CUDNNWINAPI kunetConvolutionBackwardBias(   hipdnnHandle_t                   handle,
                                                          const void                     *alpha,
                                                          const hipdnnTensorDescriptor_t   srcDesc,
                                                          const void                      *srcData,
                                                          const void                      *beta,
                                                          const hipdnnTensorDescriptor_t   destDesc,
                                                          void                           *destData
                                                      ){
    return HIPDNN_STATUS_NOT_SUPPORTED;
}

hipdnnStatus_t CUDNNWINAPI kunetPoolingForward(  hipdnnHandle_t handle,
                                                const hipdnnPoolingDescriptor_t   poolingDesc,
                                                const void                      *alpha,
                                                const hipdnnTensorDescriptor_t    srcDesc,
                                                const void                      *srcData,
                                                const void                      *beta,
                                                const hipdnnTensorDescriptor_t    destDesc,
                                                void                            *destData
                                             ){
    return HIPDNN_STATUS_NOT_SUPPORTED;
}

hipdnnStatus_t CUDNNWINAPI kunetPoolingBackward( hipdnnHandle_t                   handle,
                                                const hipdnnPoolingDescriptor_t  poolingDesc,
                                                const void                      *alpha,
                                                const hipdnnTensorDescriptor_t   srcDesc,
                                                const void                     *srcData,
                                                const hipdnnTensorDescriptor_t   srcDiffDesc,
                                                const void                     *srcDiffData,
                                                const hipdnnTensorDescriptor_t   destDesc,
                                                const void                     *destData,
                                                const void                     *beta,
                                                const hipdnnTensorDescriptor_t   destDiffDesc,
                                                void                           *destDiffData
                                              ){
    return HIPDNN_STATUS_NOT_SUPPORTED;
}
