#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include <math.h>
#include "kudnn.h"
#include <time.h>
#include "test.h"
#include <string.h>
#include <sys/time.h>

static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
        //cerr << "ERROR: Bad call to gettimeofday" << endl;
        printf("ERROR: Bad call to gettimeofday\n");
        return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

int main(int argc, char *argv[]){
    int CONV, VERBOSE;
    if(argc==3){
        if(!strcmp(argv[1], "conv"))
            CONV=1;
        else if(!strcmp(argv[1],"xcorr"))
            CONV=0;
        else{
            printf("usage: ./testConv4d xcorr/conv v[01]\n"); exit(-1);
        }

        if(!strcmp(argv[2], "v0"))
            VERBOSE=0;
        else if(!strcmp(argv[2],"v1"))
            VERBOSE=1;
        else{
            printf("usage: ./testConv4d xcorr/conv v[01]\n"); exit(-1);
        }
    }else{
        printf("usage: ./testConv4d xcorr/conv v[01]\n"); exit(-1);
    }

    int N, C, H, W; // src
    int K, Hw, Ww; // flt
    if(VERBOSE){
        N=1; C=1; H=5; W=4; // src
        K=1; Hw=2; Ww=2; // flt
    }else{
        /*
        N=28; C=3; H=40; W=80; // src
        K=5; Hw=8; Ww=7; // flt
        */
        N=128; C=3; H=40; W=80; // src
        K=5; Hw=18; Ww=17; // flt
    }

    int convHpad=0, convWpad=0, convHSt=1, convWSt=1;

    assert(H>=Hw); assert(W>=Ww);
    srand(time(NULL));
    const int Hy=1+(H+2*convHpad-Hw)/convHSt, Wy=1+(W+2*convWpad-Ww)/convWSt; // dst 
    double xData[N*C*H*W]; fillRandom(xData,N*C*H*W);
    double wData[K*C*Hw*Hw]; fillRandom(wData, K*C*Hw*Ww);
    double dyData[N*K*Hy*Wy]; fillRandom(dyData, N*K*Hy*Wy);

    double t0, time_elapsed;

    printf("N:%d C:%d H:%d W:%d\n",N,C,H,W);
    printf("K:%d C:%d Hw:%d Ww:%d\n",K,C,Hw,Ww);
    printf("N:%d K:%d Hy:%d Wy:%d\n",N,K,Hy,Wy);
    printf("\n");

    if(VERBOSE){
        printf("x:\n");
        print2Dd(xData, H, W);
        printf("w:\n");
        print2Dd(wData, Hw, Ww);
        printf("dy:\n");
        print2Dd(dyData, Hy, Wy);
        printf("\n");
    }

    double *x_h = &xData[0], *w_h = &wData[0], *dy_h=&dyData[0]; // given
    double dx_h[N*C*H*W], dw_h[C*K*Hw*Ww], y_h[N*K*Hy*Wy], db_h[1*K*1*1]; // compute cudnn
    double dx1_h[N*C*H*W], dw1_h[K*C*Hw*Ww], y1_h[N*K*Hy*Wy], db1_h[1*K*1*1]; // compute kunet
    double *x_d=NULL, *dx_d, *w_d, *dw_d, *y_d, *dy_d, *db_d; // gpu pointers

    gpuErrchk( hipMalloc(&x_d,     sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&dx_d,    sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&w_d,     sizeof(double)*K*C*Hw*Ww) );
    gpuErrchk( hipMalloc(&dw_d,    sizeof(double)*K*C*Hw*Ww) );
    gpuErrchk( hipMalloc(&y_d,     sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&dy_d,    sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&db_d,    sizeof(double)*1*K*1*1) );


    // send x, w, dy to GPU
    gpuErrchk( hipMemcpy(x_d, x_h, sizeof(double)*N*C*H*W, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(w_d, w_h, sizeof(double)*K*C*Hw*Ww, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d, dy_h, sizeof(double)*N*K*Hy*Wy, hipMemcpyHostToDevice) );
    // end send x, w, dy to GPU
    
    /**
      CUDNN KUNET COMPARISON TESTS
    **/
    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnTensorDescriptor_t         dbDesc = NULL;
    hipdnnFilterDescriptor_t         wDesc = NULL;
    hipdnnFilterDescriptor_t         dwDesc = NULL;
    hipdnnConvolutionDescriptor_t    xcorr00Desc = NULL;
    hipdnnConvolutionDescriptor_t    conv00Desc = NULL;
    hipdnnConvolutionDescriptor_t    xcorrppDesc = NULL;


    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dbDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &wDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &dwDesc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &xcorr00Desc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &xcorrppDesc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &conv00Desc) );
    // end creation

    // set
    cudnnErrchk( hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetFilter4dDescriptor(wDesc, HIPDNN_DATA_DOUBLE, K, C, Hw, Ww) );
    cudnnErrchk( hipdnnSetFilter4dDescriptor(dwDesc, HIPDNN_DATA_DOUBLE, K, C, Hw, Ww) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dbDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, 1, 1) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(xcorr00Desc, 0,0,1,1,1,1, HIPDNN_CROSS_CORRELATION) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(xcorrppDesc, convHpad,convWpad,convHSt,convWSt,1,1, HIPDNN_CROSS_CORRELATION) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(conv00Desc, 0,0,1,1,1,1, HIPDNN_CONVOLUTION) );
    // end set input and conf

    // set conv mode
    hipdnnConvolutionDescriptor_t    tconvDesc = NULL;
    if(CONV){
        tconvDesc = conv00Desc;
        printf("mode: conv00\n");
    }else{
        // tconvDesc = xcorr00Desc;
        tconvDesc = xcorrppDesc;
        printf("mode: xcorrpp\n");
    }
    // end set conv mode

    // forward algo conf & workspace
    double alpha=1, beta=1; //scaling params for input and output
    hipdnnConvolutionFwdAlgo_t convFwdAlgo;
    hipdnnConvolutionFwdPreference_t convFwdPref = HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE;
    void *workSpace = NULL; size_t workSpaceSize = 0, memLimit=0;
    cudnnErrchk( hipdnnGetConvolutionForwardAlgorithm(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdPref, memLimit, &convFwdAlgo) );
    cudnnErrchk( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdAlgo, &workSpaceSize) );
    //printf("workspace size: %d\n", workSpaceSize);
    // end forward algo conf & workspace

    // forward test
    printf("\ny:\n");
    t0 = getTime();
    cudnnErrchk( hipdnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipPeekAtLastError() ); gpuErrchk( hipDeviceSynchronize() );
    time_elapsed = getTime() - t0; printf("cudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );

    t0 = getTime();
    cudnnErrchk( kunetConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    time_elapsed = getTime() - t0; printf("kudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(y1_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(y_h, Hy, Wy); printf("\n"); print2Dd(y1_h, Hy, Wy);}
    assert(eqseq(y_h,y1_h,N*K*Hy*Wy) < 1.0E-4);
    printf("y: ok.\n");
    // end forward test

    // backward filter test
    printf("\ndw:\n");
    t0 = getTime();
    cudnnErrchk( hipdnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipPeekAtLastError() ); gpuErrchk( hipDeviceSynchronize() );
    time_elapsed = getTime() - t0; printf("cudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(dw_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );

    t0 = getTime();
    cudnnErrchk( kunetConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    time_elapsed = getTime() - t0; printf("kudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(dw1_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(dw_h, Hw, Ww); printf("\n"); print2Dd(dw1_h, Hw, Ww);}
    assert(eqseq(dw_h,dw1_h,K*C*Hw*Ww) < 1.0E-4);
    printf("dw: ok.\n");
    //print2Dd(dw_h, Hw, Ww); printf("\n");
    // end backward filter test

    // backward data test
    printf("\ndx:\n");
    t0 = getTime();
    cudnnErrchk( hipdnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipPeekAtLastError() ); gpuErrchk( hipDeviceSynchronize() );
    time_elapsed = getTime() - t0; printf("cudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );

    t0 = getTime();
    cudnnErrchk( kunetConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    time_elapsed = getTime() - t0; printf("kudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(dx1_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(dx_h, H, W); printf("\n");print2Dd(dx1_h, H, W);}
    assert(eqseq(dx_h,dx1_h,N*C*H*W) < 1.0E-4);
    printf("dx: ok.\n");
    // end backward data test

    // backward bias test
    printf("\ndb:\n");
    t0 = getTime();
    cudnnErrchk( hipdnnConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipPeekAtLastError() ); gpuErrchk( hipDeviceSynchronize() );
    time_elapsed = getTime() - t0; printf("cudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(db_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );

    t0 = getTime();
    cudnnErrchk( kunetConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    time_elapsed = getTime() - t0; printf("kudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(db1_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(db_h, 1, K); printf("\n");print2Dd(db1_h, 1, K);}
    assert(eqseq(db_h,db1_h,1*K*1*1) < 1.0E-4);
    printf("db: ok.\n\n");
    // end backward bias test

    printf("ok.\n");

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (wDesc != NULL) hipdnnDestroyFilterDescriptor(wDesc);
    if (dwDesc != NULL) hipdnnDestroyFilterDescriptor(dwDesc);
    if (yDesc != NULL) hipdnnDestroyTensorDescriptor(yDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (dbDesc != NULL) hipdnnDestroyTensorDescriptor(dbDesc);
    if (xcorr00Desc != NULL) hipdnnDestroyConvolutionDescriptor(xcorr00Desc);
    if (xcorrppDesc != NULL) hipdnnDestroyConvolutionDescriptor(xcorrppDesc);
    if (conv00Desc != NULL) hipdnnDestroyConvolutionDescriptor(conv00Desc);
    if (handle != NULL) hipdnnDestroy(handle);

    // free
    hipFree(x_d); hipFree(dx_d); hipFree(w_d); hipFree(dw_d); hipFree(y_d); hipFree(dy_d); hipFree(db_d);
    // END TESTS
    return 0;
}

