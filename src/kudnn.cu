#include <stdio.h>
#include <assert.h>
#include "kuassert.h"
#include "kudnn.h"
#include "util.h"
#include "xcorr.cuh"
#include "pool.cuh"

#define BLK 4096
#define THR 256

hipdnnStatus_t CUDNNWINAPI kudnnConvolutionForward(        hipdnnHandle_t                     handle,
                                                          const void                         *alpha,
                                                          const hipdnnTensorDescriptor_t       srcDesc,
                                                          const void                         *src,
                                                          const hipdnnFilterDescriptor_t       filterDesc,
                                                          const void                         *flt,
                                                          const hipdnnConvolutionDescriptor_t  convDesc,
                                                          hipdnnConvolutionFwdAlgo_t           algo,
                                                          void                               *workSpace,
                                                          size_t                              workSpaceSizeInBytes,            
                                                          const void                         *beta,
                                                          const hipdnnTensorDescriptor_t       destDesc,
                                                          void                               *dst
                                                 ){
    hipdnnDataType_t dataType;
    hipdnnConvolutionMode_t mode;
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

    int i;
    int ndimsreq=5; int convndimsreq=3;
    int wndims, wDims[ndimsreq];
    int xndims, xDims[ndimsreq], xStrides[ndimsreq];
    int yndims, yDims[ndimsreq], yStrides[ndimsreq];
    int convndims, convPad[convndimsreq], convStride[convndimsreq], convUpscale[convndimsreq];

    // x
    hipdnnGetTensorNdDescriptor(srcDesc,  ndimsreq, &dataType, &xndims, xDims, xStrides);

    // w
    hipdnnTensorFormat_t tensor_format;
    hipdnnGetFilterNdDescriptor(filterDesc, ndimsreq, &dataType, &tensor_format,  &wndims, wDims);
    assert(xndims == wndims);

    // y
    hipdnnGetTensorNdDescriptor(destDesc,  ndimsreq, &dataType, &yndims, yDims, yStrides);
    assert(xndims == yndims);

    hipdnnDataType_t dtype;
    cudnnGetConvolutionNdDescriptor(convDesc, convndimsreq, &convndims, convPad, convStride, convUpscale, &mode, &dtype);
    assert(convndims==(xndims-2)); for(i=0; i<convndims; i++) assert(convStride[i]==1); for(i=0; i<convndims; i++) assert(convUpscale[i]==1);

    if(xndims == 4){ // 4-D
        xDims[4] = 1; wDims[4] = 1; yDims[4] = 1; 
        xStrides[4]=1; yStrides[4]=1;
        convPad[2] = 0; convStride[2] = 0; convUpscale[2] = 0;
    }

    if(mode == HIPDNN_CROSS_CORRELATION){
        // xcorr(x,w)
        krnlXCorrY5d<<<BLK, THR>>>(
                (double *)src, cat5d(xDims),
                (double *)flt, cat5d(wDims),
                (double *)dst, cat5d(yDims),
                cat5d(yStrides), cat3d(convPad), prod5d(yDims));
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

    }else if(mode == HIPDNN_CONVOLUTION){
        // conv(x,w)
        status = HIPDNN_STATUS_NOT_SUPPORTED;
    }else{
        status = HIPDNN_STATUS_BAD_PARAM;
    }
    return status;
}

hipdnnStatus_t CUDNNWINAPI kudnnConvolutionBackwardFilter( hipdnnHandle_t                       handle,
                                                          const void                         *alpha,
                                                          const hipdnnTensorDescriptor_t       srcDesc,
                                                          const void                         *src,
                                                          const hipdnnTensorDescriptor_t       diffDesc,
                                                          const void                         *dff,
                                                          const hipdnnConvolutionDescriptor_t  convDesc,
                                                          const void                         *beta,
                                                          const hipdnnFilterDescriptor_t       gradDesc,
                                                          void                               *grd
                                                        ){
    // dw = xcorr(x,dy)
    hipdnnDataType_t dataType;
    hipdnnConvolutionMode_t mode;
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

    int i;
    int ndimsreq=5; int convndimsreq=3;
    int dwndims, dwDims[ndimsreq];
    int xndims, xDims[ndimsreq], xStrides[ndimsreq];
    int dyndims, dyDims[ndimsreq], dyStrides[ndimsreq];
    int convndims, convPad[convndimsreq], convStride[convndimsreq], convUpscale[convndimsreq];

    // x
    hipdnnGetTensorNdDescriptor(srcDesc,  ndimsreq, &dataType, &xndims, xDims, xStrides);

    // dy
    hipdnnGetTensorNdDescriptor(diffDesc,  ndimsreq, &dataType, &dyndims, dyDims, dyStrides);
    assert(xndims == dyndims);

    // dw
    hipdnnTensorFormat_t tensor_format;
    hipdnnGetFilterNdDescriptor(gradDesc, ndimsreq, &dataType, &tensor_format, &dwndims, dwDims);
    assert(xndims == dwndims);

    hipdnnDataType_t dtype;
    cudnnGetConvolutionNdDescriptor(convDesc, convndimsreq, &convndims, convPad, convStride, convUpscale, &mode, &dtype);
    assert(convndims==(xndims-2)); for(i=0; i<convndims; i++) assert(convStride[i]==1); for(i=0; i<convndims; i++) assert(convUpscale[i]==1);

    if(xndims == 4){ // 4-D
        xDims[4] = 1; dwDims[4] = 1; dyDims[4] = 1; 
        xStrides[4]=1; dyStrides[4]=1;
        convPad[2] = 0; convStride[2] = 0; convUpscale[2] = 0;
    }

    if(mode == HIPDNN_CROSS_CORRELATION){
        // dw = xcorr(x,dy)
        krnlXCorrDw5d<<<BLK, THR>>>(
                (double *)src, cat5d(xDims),
                (double *)dff, cat5d(dyDims),
                (double *)grd, cat5d(dwDims),
                dims2strides5d(dwDims), cat3d(convPad), prod5d(dwDims));
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

    }else if(mode == HIPDNN_CONVOLUTION){
        status = HIPDNN_STATUS_NOT_SUPPORTED;
    }else{
        status = HIPDNN_STATUS_BAD_PARAM;
    }
    return status;
}

hipdnnStatus_t CUDNNWINAPI kudnnConvolutionBackwardData(  hipdnnHandle_t                       handle,
                                                         const void                         *alpha,
                                                         const hipdnnFilterDescriptor_t       filterDesc,
                                                         const void                         *flt,
                                                         const hipdnnTensorDescriptor_t       diffDesc,
                                                         const void                         *dff,
                                                         const hipdnnConvolutionDescriptor_t  convDesc,
                                                         const void                         *beta,
                                                         const hipdnnTensorDescriptor_t       gradDesc,
                                                         void                               *grd
                                                       ){

    // conv(dy,w,'full');
    hipdnnDataType_t dataType;
    hipdnnConvolutionMode_t mode;
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

    int i;
    int ndimsreq=5; int convndimsreq=3;
    int dyndims, dyDims[ndimsreq], dyStrides[ndimsreq];
    int wndims, wDims[ndimsreq];
    int dxndims, dxDims[ndimsreq], dxStrides[ndimsreq];
    int convndims, convPad[convndimsreq], convStride[convndimsreq], convUpscale[convndimsreq];

    // dy
    hipdnnGetTensorNdDescriptor(diffDesc,  ndimsreq, &dataType, &dyndims, dyDims, dyStrides);

    // w
    hipdnnTensorFormat_t tensor_format;
    hipdnnGetFilterNdDescriptor(filterDesc, ndimsreq, &dataType, &tensor_format, &wndims, wDims);

    // dx
    hipdnnGetTensorNdDescriptor(gradDesc,  ndimsreq, &dataType, &dxndims, dxDims, dxStrides);
    assert(dxndims == dyndims); assert(dxndims == wndims);

    cudnnGetConvolutionNdDescriptor(convDesc, convndimsreq, &convndims, convPad, convStride, convUpscale, &mode, &dataType);
    assert(convndims==(dxndims-2)); for(i=0; i<convndims; i++) assert(convStride[i]==1); for(i=0; i<convndims; i++) assert(convUpscale[i]==1);

    if(dxndims == 4){ // 4-D
        dxDims[4] = 1; wDims[4] = 1; dyDims[4] = 1; 
        dxStrides[4]=1; dyStrides[4]=1;
        convPad[2] = 0; convStride[2] = 0; convUpscale[2] = 0;
    }

    if(mode == HIPDNN_CROSS_CORRELATION){
        // conv(dy,w,'full');
        krnlXCorrDx5d<<<BLK, THR>>>(
                (double *)dff, cat5d(dyDims),
                (double *)flt, cat5d(wDims),
                (double *)grd, cat5d(dxDims),
                cat5d(dxStrides), cat3d(convPad), prod5d(dxDims));
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

    }else if(mode == HIPDNN_CONVOLUTION){
        status = HIPDNN_STATUS_NOT_SUPPORTED;
    }else{
        status = HIPDNN_STATUS_BAD_PARAM;
    }
    return status;
}

hipdnnStatus_t CUDNNWINAPI kudnnConvolutionBackwardBias(   hipdnnHandle_t                   handle,
                                                          const void                     *alpha,
                                                          const hipdnnTensorDescriptor_t   srcDesc,
                                                          const void                      *srcData,
                                                          const void                      *beta,
                                                          const hipdnnTensorDescriptor_t   destDesc,
                                                          void                           *destData
                                                      ){
    // dy -> db : N,K,Hy,Wy,Dy -> 1,K,1,1,1
    hipdnnDataType_t dataType;
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

    int ndimsreq=5;
    int dyndims, dyDims[ndimsreq], dyStrides[ndimsreq];

    // dy
    hipdnnGetTensorNdDescriptor(srcDesc,  ndimsreq, &dataType, &dyndims, dyDims, dyStrides);

    if(dyndims == 4){ // 4-D
        dyDims[4] = 1; 
        dyStrides[4]=1;
    }

    dim3 threads(dyDims[1], 1, 1); 
    dim3 grid(1,1,1);
    krnlBackBias5d<<<grid,threads>>>(
            (double *)srcData, 
            cat5d(dyDims),
            (double *)destData
            );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    return status;
}

hipdnnStatus_t CUDNNWINAPI kudnnPoolingForward(  hipdnnHandle_t handle,
                                                const hipdnnPoolingDescriptor_t   poolingDesc,
                                                const void                      *alpha,
                                                const hipdnnTensorDescriptor_t    srcDesc,
                                                const void                      *src,
                                                const void                      *beta,
                                                const hipdnnTensorDescriptor_t    destDesc,
                                                void                            *dst
                                             ){
    hipdnnPoolingMode_t mode;
    hipdnnDataType_t dataType;
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

    int i;
    int ndimsreq=5, poolndimsreq=3, poolndims;
    int xndims, xDims[ndimsreq], xStrides[ndimsreq];
    int yndims, yDims[ndimsreq], yStrides[ndimsreq];
    int poolDims[poolndimsreq], poolPad[poolndimsreq], poolStride[poolndimsreq];

    // x
    hipdnnGetTensorNdDescriptor(srcDesc,  ndimsreq, &dataType, &xndims, xDims, xStrides);

    // y
    hipdnnGetTensorNdDescriptor(destDesc,  ndimsreq, &dataType, &yndims, yDims, yStrides);
    assert(xndims == yndims);

    hipdnnNanPropagation_t nan_prop;
    cudnnGetPoolingNdDescriptor(poolingDesc, poolndimsreq, &mode, &nan_prop, &poolndims, poolDims, poolPad, poolStride);
    for(i=0;i<poolndims;i++) assert(poolDims[i]>=poolStride[i]);

    if(xndims == 4){ // 4-D
        xDims[4] = 1; yDims[4] = 1; 
        xStrides[4]=1; yStrides[4]=1;
        poolDims[2] = 1; poolPad[2] = 0; poolStride[2] = 0;
    }

    if(mode == HIPDNN_POOLING_MAX){
            krnlMaxPoolY5d<<<BLK,THR>>>(  
                    (double *)src,
                    cat5d(xDims),
                    cat3d(poolDims),
                    cat3d(poolStride),
                    (double *)dst,
                    cat5d(yDims),
                    cat5d(yStrides),
                    prod5d(yDims)
                    );
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
    }else{
        status = HIPDNN_STATUS_NOT_SUPPORTED;
    }
    return status;
}

hipdnnStatus_t CUDNNWINAPI kudnnPoolingBackward( hipdnnHandle_t                   handle,
                                                const hipdnnPoolingDescriptor_t  poolingDesc,
                                                const void                      *alpha,
                                                const hipdnnTensorDescriptor_t   srcDesc,
                                                const void                     *srcData,
                                                const hipdnnTensorDescriptor_t   srcDiffDesc,
                                                const void                     *srcDiffData,
                                                const hipdnnTensorDescriptor_t   destDesc,
                                                const void                     *destData,
                                                const void                     *beta,
                                                const hipdnnTensorDescriptor_t   destDiffDesc,
                                                void                           *destDiffData
                                              ){
    hipdnnPoolingMode_t mode;
    hipdnnDataType_t dataType;
    hipdnnStatus_t status = HIPDNN_STATUS_SUCCESS;

    int i;
    int ndimsreq=5, poolndimsreq=3, poolndims;
    int xndims, xDims[ndimsreq], xStrides[ndimsreq];
    int dxndims, dxDims[ndimsreq], dxStrides[ndimsreq];
    int yndims, yDims[ndimsreq], yStrides[ndimsreq];
    int dyndims, dyDims[ndimsreq], dyStrides[ndimsreq];
    int poolDims[poolndimsreq], poolPad[poolndimsreq], poolStride[poolndimsreq];

    // y
    hipdnnGetTensorNdDescriptor(srcDesc,  ndimsreq, &dataType, &yndims, yDims, yStrides);

    // dy
    hipdnnGetTensorNdDescriptor(srcDiffDesc,  ndimsreq, &dataType, &dyndims, dyDims, dyStrides);

    // x
    hipdnnGetTensorNdDescriptor(destDesc,  ndimsreq, &dataType, &xndims, xDims, xStrides);

    // dx
    hipdnnGetTensorNdDescriptor(destDiffDesc,  ndimsreq, &dataType, &dxndims, dxDims, dxStrides);

    hipdnnNanPropagation_t nan_prop;
    cudnnGetPoolingNdDescriptor(poolingDesc, poolndimsreq, &mode, &nan_prop, &poolndims, poolDims, poolPad, poolStride);
    for(i=0;i<poolndims;i++) assert(poolDims[i]>=poolStride[i]);

    if(xndims == 4){ // 4-D
        xDims[4] = 1; yDims[4] = 1; 
        xStrides[4]=1; yStrides[4]=1;
        poolDims[2] = 1; poolPad[2] = 0; poolStride[2] = 0;
    }

    if(mode == HIPDNN_POOLING_MAX){
        krnlMaxPool5dDx<<<BLK,THR>>>( 
                (double *)srcData,
                cat5d(yDims),
                (double *)srcDiffData,
                (double *)destData,
                cat5d(xDims),
                (double *)destDiffData,
                cat3d(poolDims),
                cat3d(poolStride),
                cat5d(yStrides),
                prod5d(yDims)
                );
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }else{
        status = HIPDNN_STATUS_NOT_SUPPORTED;
    }

    return status;
}
