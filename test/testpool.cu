#include <stdlib.h>
#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include "../src/kudnn.h"
#include <time.h>
#include <string.h>
#include "testutil.h"
#include "../src/kuassert.h"
#include "../src/kudnn.h"



void testPooling(
        int tdims, int xDims[],
        int pdims, int poolDims[], int poolPad[], int poolStride[],
        int verbose, int compare
    ){
    int i;
    int yDims[tdims], xStrides[tdims], yStrides[tdims];

    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnPoolingDescriptor_t        poolDesc = NULL;

    // create
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreatePoolingDescriptor(      &poolDesc) );
    // end create

    // set
    // cudnnErrchk( hipdnnSetPoolingNdDescriptor(poolDesc, HIPDNN_POOLING_MAX, pdims, poolDims, poolPad, poolStride) );
    cudnnErrchk( hipdnnSetPoolingNdDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN, pdims, poolDims, poolPad, poolStride) );

    dims2strides(xDims, tdims, xStrides);
    cudnnErrchk( hipdnnSetTensorNdDescriptor(xDesc, HIPDNN_DATA_DOUBLE, tdims, xDims, xStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dxDesc, HIPDNN_DATA_DOUBLE, tdims, xDims, xStrides) );

    getPoolingNdForwardOutputDim(xDims, pdims, poolDims, poolPad, poolStride, yDims);
    dims2strides(yDims, tdims, yStrides);
    cudnnErrchk( hipdnnSetTensorNdDescriptor(yDesc, HIPDNN_DATA_DOUBLE, tdims, yDims, yStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dyDesc, HIPDNN_DATA_DOUBLE, tdims, yDims, yStrides) );
    // end set

    printf("x:\t"); for(i=0;i<tdims;i++) printf("%d\t", xDims[i]); printf("\n");
    printf("w:\t"); printf("\t\t"); for(i=0;i<pdims;i++) printf("%d\t", poolDims[i]); printf("\tpool window\n");
    printf("p:\t"); printf("\t\t"); for(i=0;i<pdims;i++) printf("%d\t", poolPad[i]); printf("\tpadding\n");
    printf("s:\t"); printf("\t\t"); for(i=0;i<pdims;i++) printf("%d\t", poolStride[i]); printf("\tstride\n");
    printf("y:\t"); for(i=0;i<tdims;i++) printf("%d\t", yDims[i]); printf("\n");

    // random data
    double xData[prod(xDims,tdims)];    fillRandom(xData,prod(xDims,tdims));
    double dyData[prod(yDims,tdims)];   fillRandom(dyData,prod(yDims,tdims));
    // end random data

    double *x_h = &xData[0], *dy_h = &dyData[0]; // given
    double y_h[prod(yDims,tdims)], dx_h[prod(xDims,tdims)]; // compute kudnn
    double y1_h[prod(yDims,tdims)], dx1_h[prod(xDims,tdims)]; // compute cudnn
    // gpu pointers
    double *x_d, *dy_d;
    double *y_d, *dx_d; 
    double *y1_d, *dx1_d; 

    // send x, dy to GPU
    gpuErrchk( hipMalloc(&x_d,     sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMalloc(&dy_d,    sizeof(double)*prod(yDims,tdims)) );

    gpuErrchk( hipMemcpy(x_d, x_h, sizeof(double)*prod(xDims,tdims), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d, dy_h, sizeof(double)*prod(yDims,tdims), hipMemcpyHostToDevice) );
    // end send x, dy to GPU

    // y, dx
    gpuErrchk( hipMalloc(&dx_d,    sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMalloc(&dx1_d,   sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMalloc(&y_d,     sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMalloc(&y1_d,    sizeof(double)*prod(yDims,tdims)) );

    // memset!
    gpuErrchk( hipMemset(y_d, 0,       sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMemset(dx_d, 0,      sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMemset(y1_d, 0,      sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMemset(dx1_d, 0,     sizeof(double)*prod(xDims,tdims)) );


    // forward test
    double alpha=1, beta=1;
    printf("y:\n");
    cudnnErrchk( kudnnPoolingForward(handle, poolDesc, &alpha, xDesc, x_d, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*prod(yDims,tdims), hipMemcpyDeviceToHost) );
    if(verbose){ print2Dd(y_h, yDims[2], yDims[3]); printf("\n");} 

    if(compare){
        cudnnErrchk( hipdnnPoolingForward(handle, poolDesc, &alpha, xDesc, x_d, &beta, yDesc, y1_d) );
        gpuErrchk( hipMemcpy(y1_h, y1_d, sizeof(double)*prod(yDims,tdims), hipMemcpyDeviceToHost) );
        if(verbose){ print2Dd(y1_h, yDims[2], yDims[3]); printf("\n");} 
        assert(eqseq(y_h,y1_h,prod(yDims,tdims)) < 1.0E-4);
    }
    printf("y: ok.\n\n");
    // end forward test 

    // backward test
    printf("dx:\n");
    cudnnErrchk( kudnnPoolingBackward(handle, poolDesc, &alpha, yDesc, y_d, dyDesc, dy_d, xDesc, x_d, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*prod(xDims,tdims), hipMemcpyDeviceToHost) );
    if(verbose){ print2Dd(dx_h, xDims[2], xDims[3]); printf("\n");} 

    if(compare){
        cudnnErrchk( hipdnnPoolingBackward(handle, poolDesc, &alpha, yDesc, y_d, dyDesc, dy_d, xDesc, x_d, &beta, dxDesc, dx1_d) );
        gpuErrchk( hipMemcpy(dx1_h, dx1_d, sizeof(double)*prod(xDims,tdims), hipMemcpyDeviceToHost) );
        if(verbose){ print2Dd(dx1_h, xDims[2], xDims[3]); printf("\n");} 
        assert(eqseq(dx_h,dx1_h,prod(xDims,tdims)) < 1.0E-4);
    }
    printf("dx:ok\n");
    // end backward test

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (poolDesc != NULL) hipdnnDestroyPoolingDescriptor(poolDesc);
    if (handle != NULL) hipdnnDestroy(handle);
    // end destroy

    // free
    hipFree(x_d); hipFree(dy_d);
    hipFree(y_d); hipFree(dx_d); 
    hipFree(y1_d); hipFree(dx1_d); 
    // end free
}

