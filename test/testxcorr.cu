#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <hipDNN.h>
#include "../src/kudnn.h"
#include <time.h>
#include "testutil.h"
#include "../src/kuassert.h"
#include "../src/kudnn.h"



void testXcorr(
        int tdims, int xDims[], int wDims[],
        int cdims, int convPad[], int convStride[], int convUpscale[], int verbose, int compare
        ){

    int i; 
    for(i=2;i<tdims;i++) assert(xDims[i]>=wDims[i]);
    for(i=0;i<cdims;i++) {assert(convUpscale[i]==1); assert(convStride[i]==1);}

    int K=wDims[0];
    int yDims[tdims], xStrides[tdims], yStrides[tdims], dbStrides[tdims]; 
    int dbDims[tdims]; for(i=0;i<tdims;i++){ dbDims[i]=1; } dbDims[1]=K;

    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnTensorDescriptor_t         dbDesc = NULL;
    hipdnnFilterDescriptor_t         wDesc = NULL;
    hipdnnFilterDescriptor_t         dwDesc = NULL;
    hipdnnConvolutionDescriptor_t    convDesc = NULL;


    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dbDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &wDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &dwDesc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &convDesc) );
    // end creation

    // set
    // x, dx
    dims2strides(xDims,tdims,xStrides);
    cudnnErrchk( hipdnnSetTensorNdDescriptor(xDesc, HIPDNN_DATA_DOUBLE, tdims, xDims, xStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dxDesc, HIPDNN_DATA_DOUBLE, tdims, xDims, xStrides) );
    printf("x:\t"); for(i=0;i<tdims;i++) printf("%d\t", xDims[i]); printf("\n");

    // w, dw
    //cudnnErrchk( hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_DOUBLE, tdims, wDims) );
    cudnnErrchk( hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, tdims, wDims) );
    //cudnnErrchk( hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_DOUBLE, tdims, wDims) );
    cudnnErrchk( hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, tdims, wDims) );
    printf("w:\t"); for(i=0;i<tdims;i++) printf("%d\t", wDims[i]); printf("\n");

    // conv
    //cudnnErrchk( hipdnnSetConvolutionNdDescriptor(convDesc, cdims, convPad, convStride, convUpscale, HIPDNN_CROSS_CORRELATION) );
    cudnnErrchk( hipdnnSetConvolutionNdDescriptor(convDesc, cdims, convPad, convStride, convUpscale, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE ) );
    printf("p:\t"); printf("\t\t"); for(i=0;i<cdims;i++) printf("%d\t", convPad[i]); printf("\tpadding\n");
    printf("s:\t"); printf("\t\t"); for(i=0;i<cdims;i++) printf("%d\t", convStride[i]); printf("\tstride\n");

    // y, dy
    cudnnErrchk( cudnnGetConvolutionNdForwardOutputDim(convDesc, xDesc, wDesc, tdims, yDims) );
    printf("y:\t"); for(i=0;i<tdims;i++) printf("%d\t", yDims[i]); printf("\n");
    dims2strides(yDims,tdims,yStrides);
    cudnnErrchk( hipdnnSetTensorNdDescriptor(yDesc, HIPDNN_DATA_DOUBLE, tdims, yDims, yStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dyDesc, HIPDNN_DATA_DOUBLE, tdims, yDims, yStrides) );

    // db
    dims2strides(dbDims,tdims,dbStrides);
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dbDesc, HIPDNN_DATA_DOUBLE, tdims, dbDims, dbStrides) );
    // end set input and conf

    srand(time(NULL));
    double xData[prod(xDims,tdims)];    fillRandom(xData,   prod(xDims,tdims));
    double wData[prod(wDims,tdims)];    fillRandom(wData,   prod(wDims,tdims));
    double dyData[prod(yDims,tdims)];   fillRandom(dyData,  prod(yDims,tdims));

    double *x_h = &xData[0],            *w_h = &wData[0],           *dy_h=&dyData[0];                       // given
    double dx_h[prod(xDims,tdims)],     dw_h[prod(wDims,tdims)],    y_h[prod(yDims,tdims)],     db_h[K];    // compute kudnn
    double dx1_h[prod(xDims,tdims)],    dw1_h[prod(wDims,tdims)],   y1_h[prod(yDims,tdims)],    db1_h[K];   // compute cudnn
    double *x_d, *w_d, *dy_d; // gpu pointers
    double *y_d,    *dw_d,  *dx_d,  *db_d;       // compute kudnn
    double *y1_d,   *dw1_d, *dx1_d, *db1_d;      // compute cudnn

    if(verbose){ print2Dd(x_h, xDims[2], xDims[3]); printf("\n");} 
    if(verbose){ print2Dd(w_h, wDims[2], wDims[3]); printf("\n");} 

    gpuErrchk( hipMalloc(&x_d,     sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMalloc(&dx_d,    sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMalloc(&dx1_d,    sizeof(double)*prod(xDims,tdims)) );

    gpuErrchk( hipMalloc(&w_d,     sizeof(double)*prod(wDims,tdims)) );
    gpuErrchk( hipMalloc(&dw_d,    sizeof(double)*prod(wDims,tdims)) );
    gpuErrchk( hipMalloc(&dw1_d,    sizeof(double)*prod(wDims,tdims)) );

    gpuErrchk( hipMalloc(&y_d,     sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMalloc(&y1_d,     sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMalloc(&dy_d,    sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMalloc(&db_d,    sizeof(double)*K) );
    gpuErrchk( hipMalloc(&db1_d,    sizeof(double)*K) );

    // send x, w, dy to GPU
    gpuErrchk( hipMemcpy(x_d,      x_h,    sizeof(double)*prod(xDims,tdims), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(w_d,      w_h,    sizeof(double)*prod(wDims,tdims), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d,     dy_h,   sizeof(double)*prod(yDims,tdims), hipMemcpyHostToDevice) );
    // end send x, w, dy to GPU

    // memset!
    gpuErrchk( hipMemset(y1_d, 0,     sizeof(double)*prod(yDims,tdims)) );
    gpuErrchk( hipMemset(dx1_d, 0,     sizeof(double)*prod(xDims,tdims)) );
    gpuErrchk( hipMemset(dw1_d, 0,     sizeof(double)*prod(wDims,tdims)) );
    gpuErrchk( hipMemset(db1_d, 0,     sizeof(double)*prod(dbDims,tdims)) );
    
    // forward test

    // forward algo conf & workspace
#if (CUDNN_VERSION < 8000)
    hipdnnConvolutionFwdAlgo_t convFwdAlgo;
    hipdnnConvolutionFwdPreference_t convFwdPref = HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE;
    void *workSpace = NULL; size_t workSpaceSize = 0, memLimit=0;
    cudnnErrchk( hipdnnGetConvolutionForwardAlgorithm(handle, xDesc, wDesc, convDesc, yDesc, convFwdPref, memLimit, &convFwdAlgo) );
    cudnnErrchk( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, convDesc, yDesc, convFwdAlgo, &workSpaceSize) );
    // end forward algo conf & workspace
#else
    void *workSpace = nullptr;
    size_t workSpaceSize = 0;
    int memLimit = 0;
    hipdnnConvolutionFwdAlgoPerf_t convFwdAlgo;
    cudnnErrchk( hipdnnFindConvolutionForwardAlgorithm(handle, xDesc, wDesc, convDesc, yDesc, 1, &memLimit, &convFwdAlgo) );
    //cudnnErrchk( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, convDesc, yDesc, convFwdAlgo.algo, &workSpaceSize) );
    workSpaceSize = convFwdAlgo.memory;    
#endif
    

    printf("\ny:\n");
    double alpha=1, beta=1; //scaling params for input and output
    //cudnnErrchk( kudnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, convDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    cudnnErrchk( kudnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, convDesc, convFwdAlgo.algo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*prod(yDims,tdims), hipMemcpyDeviceToHost) );
    if(verbose){ print2Dd(y_h, yDims[2], yDims[3]); printf("\n");} 

    if(compare){
        // cudnnErrchk( hipdnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, convDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y1_d) );
	cudnnErrchk( hipdnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, convDesc, convFwdAlgo.algo, workSpace, workSpaceSize, &beta, yDesc, y1_d) );
        gpuErrchk( hipMemcpy(y1_h, y1_d, sizeof(double)*prod(yDims,tdims), hipMemcpyDeviceToHost) );
        if(verbose){ print2Dd(y1_h, yDims[2], yDims[3]); printf("\n");} 
        assert(eqseq(y_h,y1_h,prod(yDims,tdims)) < 1.0E-4);
    }
    printf("y: ok.\n");
    // end forward test

    // backward filter test
    printf("\ndw:\n");
    cudnnErrchk( kudnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, convDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipMemcpy(dw_h, dw_d, sizeof(double)*prod(wDims,tdims), hipMemcpyDeviceToHost) );
    if(verbose){ print2Dd(dw_h, wDims[2], wDims[3]); printf("\n"); }

    if(compare) {
	auto algoType = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT_TILING;    
        //cudnnErrchk( hipdnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, convDesc, &beta, dwDesc, dw1_d) );
	cudnnErrchk( hipdnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, convDesc, algoType, workSpace, workSpaceSize, &beta, dwDesc, dw1_d) );  
        gpuErrchk( hipMemcpy(dw1_h, dw1_d, sizeof(double)*prod(wDims,tdims), hipMemcpyDeviceToHost) );
        if(verbose){ print2Dd(dw1_h, wDims[2], wDims[3]); printf("\n"); }
        assert(eqseq(dw_h,dw1_h,prod(wDims,tdims)) < 1.0E-4);
    }
    printf("dw: ok.\n");
    // end backward filter test

    // backward data test
    printf("\ndx:\n");
    cudnnErrchk( kudnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, convDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*prod(xDims,tdims), hipMemcpyDeviceToHost) );
    if(verbose){ print2Dd(dx_h, xDims[2], xDims[3]); printf("\n"); }

    if(compare) {
	auto algoBwdDataType = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING;
        //cudnnErrchk( hipdnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, convDesc, &beta, dxDesc, dx1_d) );
	cudnnErrchk ( hipdnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, convDesc, algoBwdDataType, workSpace, workSpaceSize, &beta, dxDesc, dx1_d) );
        gpuErrchk( hipMemcpy(dx1_h, dx1_d, sizeof(double)*prod(xDims,tdims), hipMemcpyDeviceToHost) );
        if(verbose){ print2Dd(dx1_h, xDims[2], xDims[3]); printf("\n"); }
        assert(eqseq(dx_h,dx1_h,prod(xDims,tdims)) < 1.0E-4);
    }
    printf("dx: ok.\n");
    // end backward data test

    // backward bias test
    printf("\ndb:\n");
    cudnnErrchk( kudnnConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipMemcpy(db_h, db_d, sizeof(double)*K, hipMemcpyDeviceToHost) );
    if(verbose){ print2Dd(db_h, 1, K); printf("\n"); }

    if(compare){
        cudnnErrchk( hipdnnConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db1_d) );
        gpuErrchk( hipMemcpy(db1_h, db1_d, sizeof(double)*K, hipMemcpyDeviceToHost) );
        if(verbose){ print2Dd(db1_h, 1, K); printf("\n"); }
        assert(eqseq(db_h,db1_h,K) < 1.0E-4);
    }
    printf("db: ok.\n\n");
    // end backward bias test

    printf("ok.\n");
    /*
    */

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (wDesc != NULL) hipdnnDestroyFilterDescriptor(wDesc);
    if (dwDesc != NULL) hipdnnDestroyFilterDescriptor(dwDesc);
    if (yDesc != NULL) hipdnnDestroyTensorDescriptor(yDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (dbDesc != NULL) hipdnnDestroyTensorDescriptor(dbDesc);
    if (convDesc != NULL) hipdnnDestroyConvolutionDescriptor(convDesc);
    if (handle != NULL) hipdnnDestroy(handle);

    // free
    hipFree(x_d); hipFree(w_d); hipFree(dy_d); 
    hipFree(dx_d); hipFree(dw_d); hipFree(y_d); hipFree(db_d);
    if(compare){
        hipFree(dx1_d); hipFree(dw1_d); hipFree(y1_d); hipFree(db1_d);
    }
    // END TESTS
}
