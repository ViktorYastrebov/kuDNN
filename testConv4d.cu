#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include <math.h>
#include "kudnn.h"
#include <time.h>
#include "test.h"
#include <string.h>

int main(int argc, char *argv[]){
    int CONV;
    if(argc==2){
        if(!strcmp(argv[1], "conv"))
            CONV=1;
        else if(!strcmp(argv[1],"xcorr"))
            CONV=0;
        else
            exit(-1);
    }else{
        printf("usage: ./testConv4d <mode>\n");
        exit(-1);
    }
    srand(time(NULL));
    /*int VERBOSE=1;
    const int N=1, C=1, H=5, W=4; // src
    const int K=1, Hw=2, Ww=2; // flt*/
    int VERBOSE=0;
    const int N=28, C=3, H=12, W=13; // src
    const int K=5, Hw=5, Ww=4; // flt*/
    assert(H>=Hw); assert(W>=Ww);
    const int Hy=H-Hw+1, Wy=W-Ww+1; // dst 
    double xData[N*C*H*W]; fillRandom(xData,N*C*H*W);
    double wData[K*C*Hw*Hw]; fillRandom(wData, K*C*Hw*Ww);
    double dyData[N*K*Hy*Wy]; fillRandom(dyData, N*K*Hy*Wy);

    printf("N:%d C:%d H:%d W:%d\n",N,C,H,W);
    printf("K:%d C:%d Hw:%d Ww:%d\n",K,C,Hw,Ww);
    printf("N:%d K:%d Hy:%d Wy:%d\n",N,K,Hy,Wy);
    printf("\n");

    if(VERBOSE){
        printf("x:\n");
        print2Dd(xData, H, W);
        printf("w:\n");
        print2Dd(wData, Hw, Ww);
        printf("dy:\n");
        print2Dd(dyData, Hy, Wy);
        printf("\n");
    }

    

    double *x_h = &xData[0], *w_h = &wData[0], *dy_h=&dyData[0]; // given
    double dx_h[N*C*H*W], dw_h[C*K*Hw*Ww], y_h[N*K*Hy*Wy], db_h[1*K*1*1]; // compute cudnn
    double dx1_h[N*C*H*W], dw1_h[K*C*Hw*Ww], y1_h[N*K*Hy*Wy], db1_h[1*K*1*1]; // compute kunet
    double *x_d, *dx_d, *w_d, *dw_d, *y_d, *dy_d, *db_d; // gpu pointers

    gpuErrchk( hipMalloc(&x_d, sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&dx_d, sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&w_d, sizeof(double)*K*C*Hw*Ww) );
    gpuErrchk( hipMalloc(&dw_d, sizeof(double)*K*C*Hw*Ww) );
    gpuErrchk( hipMalloc(&y_d, sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&dy_d, sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&db_d, sizeof(double)*1*K*1*1) );

    // send x, w, dy to GPU
    gpuErrchk( hipMemcpy(x_d, x_h, sizeof(double)*N*C*H*W, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(w_d, w_h, sizeof(double)*K*C*Hw*Ww, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d, dy_h, sizeof(double)*N*K*Hy*Wy, hipMemcpyHostToDevice) );
    // end send x, w, dy to GPU
    
    /**
      CUDNN KUNET COMPARISON TESTS
    **/
    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnTensorDescriptor_t         dbDesc = NULL;
    hipdnnFilterDescriptor_t         wDesc = NULL;
    hipdnnFilterDescriptor_t         dwDesc = NULL;
    hipdnnConvolutionDescriptor_t    xcorr00Desc = NULL;
    hipdnnConvolutionDescriptor_t    conv00Desc = NULL;


    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dbDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &wDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &dwDesc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &xcorr00Desc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &conv00Desc) );
    // end creation

    // set
    cudnnErrchk( hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetFilter4dDescriptor(wDesc, HIPDNN_DATA_DOUBLE, K, C, Hw, Ww) );
    cudnnErrchk( hipdnnSetFilter4dDescriptor(dwDesc, HIPDNN_DATA_DOUBLE, K, C, Hw, Ww) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dbDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, 1, 1) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(xcorr00Desc, 0,0,1,1,1,1, HIPDNN_CROSS_CORRELATION) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(conv00Desc, 0,0,1,1,1,1, HIPDNN_CONVOLUTION) );
    // end set input and conf

    // set conv mode
    hipdnnConvolutionDescriptor_t    tconvDesc = NULL;
    if(CONV){
        tconvDesc = conv00Desc;
        printf("mode: conv00\n");
    }else{
        tconvDesc = xcorr00Desc;
        printf("mode: xcorr00\n");
    }
    // end set conv mode

    // forward algo conf & workspace
    double alpha=1, beta=1; //scaling params for input and output
    hipdnnConvolutionFwdAlgo_t convFwdAlgo;
    hipdnnConvolutionFwdPreference_t convFwdPref = HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE;
    void *workSpace = NULL; size_t workSpaceSize = 0, memLimit=0;
    cudnnErrchk( hipdnnGetConvolutionForwardAlgorithm(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdPref, memLimit, &convFwdAlgo) );
    cudnnErrchk( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdAlgo, &workSpaceSize) );
    printf("workspace size: %d\n", workSpaceSize);
    // end forward algo conf & workspace

    // forward test
    printf("\ny:\n");
    cudnnErrchk( hipdnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y1_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(y_h, Hy, Wy); printf("\n"); print2Dd(y1_h, Hy, Wy);}
    assert(eqseq(y_h,y1_h,N*K*Hy*Wy) < 1.0E-4);
    printf("y: ok.\n");
    // end forward test

    // backward filter test
    printf("\ndw:\n");
    cudnnErrchk( hipdnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipMemcpy(dw_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipMemcpy(dw1_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(dw_h, Hw, Ww); printf("\n"); print2Dd(dw1_h, Hw, Ww);}
    assert(eqseq(dw_h,dw1_h,K*C*Hw*Ww) < 1.0E-4);
    printf("dw: ok.\n");
    //print2Dd(dw_h, Hw, Ww); printf("\n");
    // end backward filter test

    // backward data test
    printf("\ndx:\n");
    cudnnErrchk( hipdnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx1_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(dx_h, H, W); printf("\n");print2Dd(dx1_h, H, W);}
    assert(eqseq(dx_h,dx1_h,N*C*H*W) < 1.0E-4);
    printf("dx: ok.\n");
    // end backward data test

    // backward bias test
    printf("\ndb:\n");
    cudnnErrchk( hipdnnConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipMemcpy(db_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipMemcpy(db1_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(db_h, 1, K); printf("\n");print2Dd(db1_h, 1, K);}
    assert(eqseq(db_h,db1_h,1*K*1*1) < 1.0E-4);
    printf("db: ok.\n\n");
    // end backward bias test

    printf("ok.\n");

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (wDesc != NULL) hipdnnDestroyFilterDescriptor(wDesc);
    if (dwDesc != NULL) hipdnnDestroyFilterDescriptor(dwDesc);
    if (yDesc != NULL) hipdnnDestroyTensorDescriptor(yDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (dbDesc != NULL) hipdnnDestroyTensorDescriptor(dbDesc);
    if (xcorr00Desc != NULL) hipdnnDestroyConvolutionDescriptor(xcorr00Desc);
    if (conv00Desc != NULL) hipdnnDestroyConvolutionDescriptor(conv00Desc);
    if (handle != NULL) hipdnnDestroy(handle);

    // free
    hipFree(x_d); hipFree(dx_d); hipFree(w_d); hipFree(dw_d); hipFree(y_d); hipFree(dy_d); hipFree(db_d);
    // END TESTS
    return 0;
}

