#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include <math.h>
#include "kudnn.h"
#include <time.h>

void  readImages(double *E, int N){
    FILE *fp;
    fp=fopen("data0", "rb");
    //size_t fread(void *ptr, size_t size_of_elements, size_t number_of_elements, FILE *a_file);
    unsigned char images[N];
    //double *E = (double*)malloc(sizeof(double)*N);
    fread(images, sizeof(unsigned char), N, fp);
    int i;
    for(i=0;i<N;i++)
        E[i] = images[i]/255.0;
    fclose(fp);
}

void fillRandom(double *E, int N){
    int i;
    for(i=0; i<N; i++)
        E[i] = rand() % 10 + 1;
}

double eqseq(double *A, double *B, int N){
    int i;
    double err=0;
    for(i=0;i<N;i++)
        err += abs(A[i]-B[i]);
    return err;
}

double xtoyData[] = 
{   1.0, 6.0, 11.0, 16.0,
    2.0, 7.0, 12.0, 17.0,
    3.0, 8.0, 13.0, 18.0,
    4.0, 9.0, 14.0, 19.0,
    5.0, 10.0, 15.0, 20.0,
    4.0, 9.0, 14.0, 19.0 };
double wtoyData[] = 
{   1.0, 3.0, 1.0, 3.0,
    2.0, 4.0, 2.0, 4.0};

int main(){
    int VERBOSE=0;
    int CONV=1;
    srand(time(NULL));
    const int N=100, C=3, H=28, W=28; // src
    const int K=10, Hw=7, Ww=7; // flt
    assert(H>=Hw); assert(W>=Ww);
    const int Hy=H-Hw+1, Wy=W-Ww+1; // dst 
    double xData[N*C*H*W]; fillRandom(xData,N*C*H*W);
    double wData[K*C*Hw*Hw]; fillRandom(wData, K*C*Hw*Ww);
    double dyData[N*K*Hy*Wy]; fillRandom(dyData, N*K*Hy*Wy);

    printf("N:%d C:%d H:%d W:%d\n",N,C,H,W);
    printf("K:%d C:%d Hw:%d Ww:%d\n",K,C,Hw,Ww);
    printf("N:%d K:%d Hy:%d Wy:%d\n",N,K,Hy,Wy);
    printf("\n");

    if(VERBOSE){
        printf("x:\n");
        print2Dd(xData, H, W);
        printf("w:\n");
        print2Dd(wData, Hw, Ww);
        printf("dy:\n");
        print2Dd(dyData, Hy, Wy);
        printf("\n");
    }

    

    double *x_h = &xData[0], *w_h = &wData[0], *dy_h=&dyData[0]; // given
    double dx_h[N*C*H*W], dw_h[C*K*Hw*Ww], y_h[N*K*Hy*Wy], db_h[1*K*1*1]; // compute cudnn
    double dx1_h[N*C*H*W], dw1_h[K*C*Hw*Ww], y1_h[N*K*Hy*Wy], db1_h[1*K*1*1]; // compute kunet
    double *x_d, *dx_d, *w_d, *dw_d, *y_d, *dy_d, *db_d; // gpu pointers

    gpuErrchk( hipMalloc(&x_d, sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&dx_d, sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&w_d, sizeof(double)*K*C*Hw*Ww) );
    gpuErrchk( hipMalloc(&dw_d, sizeof(double)*K*C*Hw*Ww) );
    gpuErrchk( hipMalloc(&y_d, sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&dy_d, sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&db_d, sizeof(double)*1*K*1*1) );

    // send x, w, dy to GPU
    gpuErrchk( hipMemcpy(x_d, x_h, sizeof(double)*N*C*H*W, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(w_d, w_h, sizeof(double)*K*C*Hw*Ww, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d, dy_h, sizeof(double)*N*K*Hy*Wy, hipMemcpyHostToDevice) );
    // end send x, w, dy to GPU
    
    /**
      CUDNN KUNET COMPARISON TESTS
    **/
    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnTensorDescriptor_t         dbDesc = NULL;
    hipdnnFilterDescriptor_t         wDesc = NULL;
    hipdnnFilterDescriptor_t         dwDesc = NULL;
    hipdnnConvolutionDescriptor_t    xcorr00Desc = NULL;
    hipdnnConvolutionDescriptor_t    conv00Desc = NULL;


    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dbDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &wDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &dwDesc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &xcorr00Desc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &conv00Desc) );
    // end creation

    // set
    cudnnErrchk( hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetFilter4dDescriptor(wDesc, HIPDNN_DATA_DOUBLE, K, C, Hw, Ww) );
    cudnnErrchk( hipdnnSetFilter4dDescriptor(dwDesc, HIPDNN_DATA_DOUBLE, K, C, Hw, Ww) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dbDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, 1, 1) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(xcorr00Desc, 0,0,1,1,1,1, HIPDNN_CROSS_CORRELATION) );
    cudnnErrchk( hipdnnSetConvolution2dDescriptor(conv00Desc, 0,0,1,1,1,1, HIPDNN_CONVOLUTION) );
    // end set input and conf

    // set conv mode
    hipdnnConvolutionDescriptor_t    tconvDesc = NULL;
    if(CONV){
        tconvDesc = conv00Desc;
        printf("mode: conv00\n");
    }else{
        tconvDesc = xcorr00Desc;
        printf("mode: xcorr00\n");
    }
    // end set conv mode

    // forward algo conf & workspace
    double alpha=1, beta=1; //scaling params for input and output
    hipdnnConvolutionFwdAlgo_t convFwdAlgo;
    hipdnnConvolutionFwdPreference_t convFwdPref = HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE;
    void *workSpace = NULL; size_t workSpaceSize = 0, memLimit=0;
    cudnnErrchk( hipdnnGetConvolutionForwardAlgorithm(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdPref, memLimit, &convFwdAlgo) );
    cudnnErrchk( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdAlgo, &workSpaceSize) );
    printf("workspace size: %d\n", workSpaceSize);
    // end forward algo conf & workspace

    // forward test
    printf("\ny:\n");
    cudnnErrchk( hipdnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y1_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(y_h, Hy, Wy); printf("\n"); print2Dd(y1_h, Hy, Wy);}
    assert(eqseq(y_h,y1_h,N*K*Hy*Wy) < 1.0E-4);
    printf("y: ok.\n");
    // end forward test

    // backward filter test
    printf("\ndw:\n");
    cudnnErrchk( hipdnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipMemcpy(dw_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipMemcpy(dw1_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(dw_h, Hw, Ww); printf("\n"); print2Dd(dw1_h, Hw, Ww);}
    assert(eqseq(dw_h,dw1_h,K*C*Hw*Ww) < 1.0E-4);
    printf("dw: ok.\n");
    //print2Dd(dw_h, Hw, Ww); printf("\n");
    // end backward filter test

    // backward data test
    printf("\ndx:\n");
    cudnnErrchk( hipdnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx1_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(dx_h, H, W); printf("\n");print2Dd(dx1_h, H, W);}
    assert(eqseq(dx_h,dx1_h,N*C*H*W) < 1.0E-4);
    printf("dx: ok.\n");
    // end backward data test

    // backward bias test
    printf("\ndb:\n");
    cudnnErrchk( hipdnnConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipMemcpy(db_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipMemcpy(db1_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(db_h, 1, K); printf("\n");print2Dd(db1_h, 1, K);}
    assert(eqseq(db_h,db1_h,1*K*1*1) < 1.0E-4);
    printf("db: ok.\n\n");
    // end backward bias test

    printf("ok.\n");

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (wDesc != NULL) hipdnnDestroyFilterDescriptor(wDesc);
    if (dwDesc != NULL) hipdnnDestroyFilterDescriptor(dwDesc);
    if (yDesc != NULL) hipdnnDestroyTensorDescriptor(yDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (dbDesc != NULL) hipdnnDestroyTensorDescriptor(dbDesc);
    if (xcorr00Desc != NULL) hipdnnDestroyConvolutionDescriptor(xcorr00Desc);
    if (conv00Desc != NULL) hipdnnDestroyConvolutionDescriptor(conv00Desc);
    if (handle != NULL) hipdnnDestroy(handle);

    // free
    hipFree(x_d); hipFree(dx_d); hipFree(w_d); hipFree(dw_d); hipFree(y_d); hipFree(dy_d); hipFree(db_d);
    // END TESTS
    return 0;
}

