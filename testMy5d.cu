#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include <math.h>
#include "kudnn.h"
#include <time.h>
#include "test.h"
#include <string.h>


void testXcorr(
        int dims,
        int N, int C, int H, int W, int D,
        int K, int Hw, int Ww, int Dw,
        int convHPad, int convWPad, int convDPad
        ){
}

void testPooling(){
    const int xDims[5] = {1,1,28,28,28}; // N C H W D

    const int poolDims[5-2] = {5,5,5};
    const int poolPad[5-2] = {0,0,0};
    const int poolStride[5-2] = {5,5,5};
    const int poolUpscale[5-2] = {1,1,1};

    const int yDims[5] = {
        xDims[0], xDims[1], // N K (C)
        1+ceil((xDims[2]+2*poolPad[0]-poolDims[0])/(double)poolStride[0]),
        1+ceil((xDims[3]+2*poolPad[1]-poolDims[1])/(double)poolStride[1]),
        1+ceil((xDims[4]+2*poolPad[2]-poolDims[2])/(double)poolStride[2])
    };

    const int xStrides[5] = { dims2strides5d(xDims) };
    const int yStrides[5] = { dims2strides5d(yDims)  };

    double xData[prod5d(xDims)]; fillRandom(xData,prod5d(xDims));
    double dyData[prod5d(yDims)]; fillRandom(dyData,prod5d(yDims));

    double *x_h = &xData[0], *dy_h = &dyData[0]; // given
    double y_h[prod5d(yDims)], dx_h[prod5d(xDims)]; // compute cudnn
    double y1_h[prod5d(yDims)], dx1_h[prod5d(xDims)]; // compute kunet
    double *x_d, *y_d, *dx_d, *dy_d; // gpu pointers

    gpuErrchk( hipMalloc(&x_d, sizeof(double)*prod5d(xDims)) );
    gpuErrchk( hipMalloc(&dx_d, sizeof(double)*prod5d(xDims)) );
    gpuErrchk( hipMalloc(&y_d, sizeof(double)*prod5d(yDims)) );
    gpuErrchk( hipMalloc(&dy_d, sizeof(double)*prod5d(yDims)) );

    // send x, dy to GPU
    gpuErrchk( hipMemcpy(x_d, x_h, sizeof(double)*prod5d(xDims), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d, dy_h, sizeof(double)*prod5d(yDims), hipMemcpyHostToDevice) );
    // end send x, dy to GPU

    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnPoolingDescriptor_t        maxPoolDesc = NULL;

    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreatePoolingDescriptor(      &maxPool00Desc) );
    // end creation

    // set
    cudnnErrchk( hipdnnSetTensorNdDescriptor(xDesc, HIPDNN_DATA_DOUBLE, 5, xDims, xStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dxDesc, HIPDNN_DATA_DOUBLE, 5, xDims, xStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(yDesc, HIPDNN_DATA_DOUBLE, 5, yDims, yStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dyDesc, HIPDNN_DATA_DOUBLE, 5, yDims, yStrides) );
    cudnnErrchk( hipdnnSetPoolingNdDescriptor(maxPoolDesc, HIPDNN_POOLING_MAX, 5-2, poolDims, poolPad, poolStride) );
    // end set input and conf
}
void poolingOutputDims(
        const int xDims[],
        const int poolDims[],
        const int poolPad[],
        const int poolStride[],
        int yDims[]){
}

int main(int argc, char *argv[]){
    int CONV;
    if(argc==2){
        if(!strcmp(argv[1], "conv"))
            CONV=1;
        else if(!strcmp(argv[1],"xcorr"))
            CONV=0;
        else
            exit(-1);
    }else{
        printf("usage: ./testConv4d <mode>\n");
        exit(-1);
    }
    srand(time(NULL));
    int ii=0; 
    int VERBOSE=0;
    const int N=100, C=3, K=10;
    const int xDims[5] = {N,C,28,28,28}; // N C H W D
    const int wDims[5] = {K,C,2,3,2}; // K C Hw Ww Dw
    for(ii=2;ii<5;ii++) assert(xDims[ii]>=wDims[ii]);

    const int convPad[5-2] = {2,2,2};
    const int convStride[5-2] = {1,1,1};
    const int convUpscale[5-2] = {1,1,1};

    const int yDims[5] = 
        {N,K,
        xDims[2]+2*convPad[0]-wDims[2]+1,
        xDims[3]+2*convPad[1]-wDims[3]+1,
        xDims[4]+2*convPad[2]-wDims[4]+1};   // N K  Hy Wy Dy; for stride=1
    const int xStrides[5] = { dims2strides5d(xDims) };
    const int yStrides[5] = { dims2strides5d(yDims)  };

    double xData[prod5d(xDims)];    fillRandom(xData,   prod5d(xDims));
    double wData[prod5d(wDims)];    fillRandom(wData,   prod5d(wDims));
    double dyData[prod5d(yDims)];   fillRandom(dyData,  prod5d(yDims));

    printf("N:%d C:%d H:%d W:%d D:%d\n",        cat5d(xDims));
    printf("K:%d C:%d Hw:%d Ww:%d Dw:%d\n",     cat5d(wDims));
    printf("N:%d K:%d Hy:%d Wy:%d Dy:%d\n",     cat5d(yDims));
    printf("conv pad: %d %d %d\n", cat3d(convPad));
    printf("conv stride: %d %d %d\n", cat3d(convStride));
    printf("\n");

    printf("strides:\n");
    printf("x: %d %d %d %d %d\n", cat5d(xStrides));
    printf("y: %d %d %d %d %d\n", cat5d(yStrides));

    printf("sizes:\n");
    printf("x: %d\n",prod5d(xDims));
    printf("w: %d\n",prod5d(wDims));
    printf("y: %d\n",prod5d(yDims));

    double *x_h = &xData[0],        *w_h = &wData[0],       *dy_h=&dyData[0];                           // given
    double dx_h[prod5d(xDims)],     dw_h[prod5d(wDims)],    y_h[prod5d(yDims)],     db_h[1*K*1*1*1];    // compute cudnn
    double dx1_h[prod5d(xDims)],    dw1_h[prod5d(wDims)],   y1_h[prod5d(yDims)],    db1_h[1*K*1*1*1];   // compute kunet
    double *x_d, *dx_d, *w_d, *dw_d, *y_d, *dy_d, *db_d; // gpu pointers

    gpuErrchk( hipMalloc(&x_d,     sizeof(double)*prod5d(xDims)) );
    gpuErrchk( hipMalloc(&dx_d,    sizeof(double)*prod5d(xDims)) );
    gpuErrchk( hipMalloc(&w_d,     sizeof(double)*prod5d(wDims)) );
    gpuErrchk( hipMalloc(&dw_d,    sizeof(double)*prod5d(wDims)) );
    gpuErrchk( hipMalloc(&y_d,     sizeof(double)*prod5d(yDims)) );
    gpuErrchk( hipMalloc(&dy_d,    sizeof(double)*prod5d(yDims)) );
    gpuErrchk( hipMalloc(&db_d,    sizeof(double)*1*K*1*1*1) );

    // send x, w, dy to GPU
    gpuErrchk( hipMemcpy(x_d,      x_h,    sizeof(double)*prod5d(xDims), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(w_d,      w_h,    sizeof(double)*prod5d(wDims), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d,     dy_h,   sizeof(double)*prod5d(yDims), hipMemcpyHostToDevice) );
    // end send x, w, dy to GPU
    
    /**
      CUDNN KUNET COMPARISON TESTS
    **/
    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnTensorDescriptor_t         dbDesc = NULL;
    hipdnnFilterDescriptor_t         wDesc = NULL;
    hipdnnFilterDescriptor_t         dwDesc = NULL;
    hipdnnConvolutionDescriptor_t    xcorr00Desc = NULL;
    hipdnnConvolutionDescriptor_t    conv00Desc = NULL;


    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dbDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &wDesc) );
    cudnnErrchk( hipdnnCreateFilterDescriptor(       &dwDesc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &xcorr00Desc) );
    cudnnErrchk( hipdnnCreateConvolutionDescriptor(  &conv00Desc) );
    // end creation

    // set
    cudnnErrchk( hipdnnSetTensorNdDescriptor(xDesc, HIPDNN_DATA_DOUBLE, 5, xDims, xStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dxDesc, HIPDNN_DATA_DOUBLE, 5, xDims, xStrides) );
    cudnnErrchk( hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_DOUBLE, 5, wDims) );
    cudnnErrchk( hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_DOUBLE, 5, wDims) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(yDesc, HIPDNN_DATA_DOUBLE, 5, yDims, yStrides) );
    cudnnErrchk( hipdnnSetTensorNdDescriptor(dyDesc, HIPDNN_DATA_DOUBLE, 5, yDims, yStrides) );
    cudnnErrchk( hipdnnSetConvolutionNdDescriptor(xcorr00Desc, 5-2, convPad, convStride, convUpscale, HIPDNN_CROSS_CORRELATION) );
    cudnnErrchk( hipdnnSetConvolutionNdDescriptor(conv00Desc, 5-2, convPad, convStride, convUpscale, HIPDNN_CONVOLUTION) );
    // end set input and conf

    // set conv mode
    hipdnnConvolutionDescriptor_t    tconvDesc = NULL;
    if(CONV){
        tconvDesc = conv00Desc;
        printf("mode: conv00\n");
    }else{
        tconvDesc = xcorr00Desc;
        printf("mode: xcorr00\n");
    }
    // end set conv mode

    // err chk
    int y1Dims[5];
    cudnnErrchk( cudnnGetConvolutionNdForwardOutputDim(tconvDesc, xDesc, wDesc, 5, y1Dims) );
    printf("N:%d K:%d Hy:%d Wy:%d Dy:%d\n",     cat5d(y1Dims));
    // end err chk

    // forward algo conf & workspace
    hipdnnConvolutionFwdAlgo_t convFwdAlgo;
    hipdnnConvolutionFwdPreference_t convFwdPref = HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE;
    void *workSpace = NULL; size_t workSpaceSize = 0, memLimit=0;
    cudnnErrchk( hipdnnGetConvolutionForwardAlgorithm(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdPref, memLimit, &convFwdAlgo) );
    cudnnErrchk( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, wDesc, tconvDesc, yDesc, convFwdAlgo, &workSpaceSize) );
    printf("workspace size: %d\n", workSpaceSize);
    // end forward algo conf & workspace

    // forward test
    printf("\ny:\n");
    double alpha=1, beta=1; //scaling params for input and output
    //cudnnErrchk( hipdnnConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    cudnnErrchk( kunetConvolutionForward(handle, &alpha, xDesc, x_d, wDesc, w_d, tconvDesc, convFwdAlgo, workSpace, workSpaceSize, &beta, yDesc, y_d) );
    gpuErrchk( hipMemcpy(y1_h, y_d, sizeof(double)*prod5d(yDims), hipMemcpyDeviceToHost) );
    /*
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    if(VERBOSE){ print2Dd(y_h, Hy, Wy); printf("\n"); print2Dd(y1_h, Hy, Wy);}
    assert(eqseq(y_h,y1_h,N*K*Hy*Wy) < 1.0E-4);
    */
    printf("y: ok.\n");
    // end forward test

    // backward filter test
    printf("\ndw:\n");
    //cudnnErrchk( hipdnnConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    //gpuErrchk( hipMemcpy(dw_h, dw_d, sizeof(double)*K*C*Hw*Ww, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardFilter(handle, &alpha, xDesc, x_d, dyDesc, dy_d, tconvDesc, &beta, dwDesc, dw_d) );
    gpuErrchk( hipMemcpy(dw1_h, dw_d, sizeof(double)*prod5d(wDims), hipMemcpyDeviceToHost) );
    /*if(VERBOSE){ print2Dd(dw_h, Hw, Ww); printf("\n"); print2Dd(dw1_h, Hw, Ww);}
    assert(eqseq(dw_h,dw1_h,K*C*Hw*Ww) < 1.0E-4);*/
    printf("dw: ok.\n");
    // end backward filter test

    // backward data test
    printf("\ndx:\n");
    // cudnnErrchk( hipdnnConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    // gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardData(handle, &alpha, wDesc, w_d, dyDesc, dy_d, tconvDesc, &beta, dxDesc, dx_d) );
    gpuErrchk( hipMemcpy(dx1_h, dx_d, sizeof(double)*prod5d(xDims), hipMemcpyDeviceToHost) );
    // if(VERBOSE){print2Dd(dx_h, H, W); printf("\n");print2Dd(dx1_h, H, W);}
    // assert(eqseq(dx_h,dx1_h,N*C*H*W) < 1.0E-4);
    printf("dx: ok.\n");
    // end backward data test

    // backward bias test
    printf("\ndb:\n");
    // cudnnErrchk( hipdnnConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    // gpuErrchk( hipMemcpy(db_h, db_d, sizeof(double)*1*K*1*1, hipMemcpyDeviceToHost) );
    cudnnErrchk( kunetConvolutionBackwardBias(handle, &alpha, dyDesc, dy_d, &beta, dbDesc, db_d) );
    gpuErrchk( hipMemcpy(db1_h, db_d, sizeof(double)*yDims[1], hipMemcpyDeviceToHost) );
    // if(VERBOSE){print2Dd(db_h, 1, K); printf("\n");print2Dd(db1_h, 1, K);}
    // assert(eqseq(db_h,db1_h,1*K*1*1) < 1.0E-4);
    printf("db: ok.\n\n");
    // end backward bias test
    /*
    */

    printf("ok.\n");

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (wDesc != NULL) hipdnnDestroyFilterDescriptor(wDesc);
    if (dwDesc != NULL) hipdnnDestroyFilterDescriptor(dwDesc);
    if (yDesc != NULL) hipdnnDestroyTensorDescriptor(yDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (dbDesc != NULL) hipdnnDestroyTensorDescriptor(dbDesc);
    if (xcorr00Desc != NULL) hipdnnDestroyConvolutionDescriptor(xcorr00Desc);
    if (conv00Desc != NULL) hipdnnDestroyConvolutionDescriptor(conv00Desc);
    if (handle != NULL) hipdnnDestroy(handle);

    // free
    hipFree(x_d); hipFree(dx_d); hipFree(w_d); hipFree(dw_d); hipFree(y_d); hipFree(dy_d); hipFree(db_d);
    // END TESTS
    return 0;
}

