#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipDNN.h>
#include <assert.h>
#include <math.h>
#include "kudnn.h"
#include <time.h>
#include "test.h"
#include <sys/time.h>

static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
        //cerr << "ERROR: Bad call to gettimeofday" << endl;
        printf("ERROR: Bad call to gettimeofday\n");
        return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()


int main(){
    int VERBOSE=0;
    //int PMODE=1; // max
    srand(time(NULL));
    const int N=28, C=3, H=40, W=80; // src
    const int K=C, Hd=8, Wd=8; // window
    const int Hs=Hd, Ws=Hd; // stride
    const int Hp=0, Wp=0; // padding
    assert(H>=Hd); assert(W>=Wd);
    const int Hy=1+ceil((H+2*Hp-Hd)/(double)Hs), Wy=1+ceil((W+2*Wp-Wd)/(double)Ws); // dst 

    printf("N:%d C:%d H:%d W:%d\n",N,C,H,W);
    printf("Hd:%d Wd:%d Hs:%d Ws:%d Hp:%d Wp:%d\n",Hd,Wd,Hs,Ws,Hp,Wp);
    printf("N:%d K:%d Hy:%d Wy:%d\n",N,C,Hy,Wy);
    printf("\n");

    double xData[N*C*H*W]; fillRandom(xData,N*C*H*W);
    double dyData[N*K*Hy*Wy]; fillRandom(dyData,N*K*Hy*Wy);

    if(VERBOSE){
        printf("x:\n");
        print2Dd(xData, H, W);
        printf("dy:\n");
        print2Dd(dyData, Hy, Wy);
        printf("\n");
    }

    double t0, time_elapsed;

    double *x_h = &xData[0], *dy_h = &dyData[0]; // given
    double y_h[N*C*Hy*Wy], dx_h[N*C*H*W]; // compute cudnn
    double y1_h[N*C*H*W], dx1_h[N*C*H*W]; // compute kunet
    double *x_d, *y_d, *dx_d, *dy_d; // gpu pointers

    gpuErrchk( hipMalloc(&x_d, sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&dx_d, sizeof(double)*N*C*H*W) );
    gpuErrchk( hipMalloc(&y_d, sizeof(double)*N*K*Hy*Wy) );
    gpuErrchk( hipMalloc(&dy_d, sizeof(double)*N*K*Hy*Wy) );

    // send x, dy to GPU
    gpuErrchk( hipMemcpy(x_d, x_h, sizeof(double)*N*C*H*W, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_d, dy_h, sizeof(double)*N*K*Hy*Wy, hipMemcpyHostToDevice) );
    // end send x, dy to GPU
    
    /**
      CUDNN KUNET COMPARISON TESTS
    **/
    hipdnnHandle_t                   handle = NULL;
    hipdnnTensorDescriptor_t         xDesc = NULL;
    hipdnnTensorDescriptor_t         dxDesc = NULL;
    hipdnnTensorDescriptor_t         yDesc = NULL;
    hipdnnTensorDescriptor_t         dyDesc = NULL;
    hipdnnPoolingDescriptor_t        maxPool00Desc = NULL;


    // creation
    cudnnErrchk( hipdnnCreate(                       &handle) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &xDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dxDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &yDesc) );
    cudnnErrchk( hipdnnCreateTensorDescriptor(       &dyDesc) );
    cudnnErrchk( hipdnnCreatePoolingDescriptor(      &maxPool00Desc) );
    // end creation

    // set
    cudnnErrchk( hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, C, H, W) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, N, K, Hy, Wy) );
    cudnnErrchk( hipdnnSetPooling2dDescriptor(maxPool00Desc, HIPDNN_POOLING_MAX, Hd,Wd,0,0,Hs,Ws) );
    // end set input and conf

    // set pool mode
    hipdnnPoolingDescriptor_t    tpoolDesc = NULL;
    tpoolDesc = maxPool00Desc;
    printf("mode: maxPool00\n");
    // end set pool mode

    double alpha=1, beta=1;

    // forward test
    printf("y:\n");
    t0 = getTime();
    cudnnErrchk( hipdnnPoolingForward(handle, tpoolDesc, &alpha, xDesc, x_d, &beta, yDesc, y_d) );
    gpuErrchk( hipPeekAtLastError() ); gpuErrchk( hipDeviceSynchronize() );
    time_elapsed = getTime() - t0; printf("cudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(y_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );

    t0 = getTime();
    cudnnErrchk( kunetPoolingForward(handle, tpoolDesc, &alpha, xDesc, x_d, &beta, yDesc, y_d) );
    time_elapsed = getTime() - t0; printf("kudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(y1_h, y_d, sizeof(double)*N*K*Hy*Wy, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(y_h, Hy, Wy); printf("\n"); print2Dd(y1_h, Hy, Wy);}
    assert(eqseq(y_h,y1_h,N*K*Hy*Wy) < 1.0E-4);
    printf("y: ok.\n\n");
    // end forward test

    // backward test
    printf("dx:\n");
    t0 = getTime();
    cudnnErrchk( hipdnnPoolingBackward(handle, tpoolDesc, &alpha, yDesc, y_d, dyDesc, dy_d, xDesc, x_d, &beta, dxDesc, dx_d) );
    gpuErrchk( hipPeekAtLastError() ); gpuErrchk( hipDeviceSynchronize() );
    time_elapsed = getTime() - t0; printf("cudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(dx_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );

    t0 = getTime();
    cudnnErrchk( kunetPoolingBackward(handle, tpoolDesc, &alpha, yDesc, y_d, dyDesc, dy_d, xDesc, x_d, &beta, dxDesc, dx_d) );
    time_elapsed = getTime() - t0; printf("kudnn: %.4f\n",time_elapsed);
    gpuErrchk( hipMemcpy(dx1_h, dx_d, sizeof(double)*N*C*H*W, hipMemcpyDeviceToHost) );
    if(VERBOSE){print2Dd(dx_h, H, W); printf("\n");print2Dd(dx1_h, H, W);}
    assert(eqseq(dx_h,dx1_h,N*C*H*W) < 1.0E-4);
    printf("dx:ok\n");
    // end backward test
    
    printf("ok.\n");

    // destroy
    if (xDesc != NULL) hipdnnDestroyTensorDescriptor(xDesc);
    if (dxDesc != NULL) hipdnnDestroyTensorDescriptor(dxDesc);
    if (dyDesc != NULL) hipdnnDestroyTensorDescriptor(dyDesc);
    if (maxPool00Desc != NULL) hipdnnDestroyPoolingDescriptor(maxPool00Desc);
    if (handle != NULL) hipdnnDestroy(handle);

    // free
    hipFree(x_d); hipFree(y_d);
    hipFree(dx_d); hipFree(dy_d);
    // END TESTS
    return 0;
}

